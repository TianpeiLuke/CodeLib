
#include <hip/hip_runtime.h>
#include <cstdlib>

int main() {
    // Allocate 3 arrays on CPU
    int nr_rows_A, nr_cols_A, nr_rows_B, nr_cols_B, nr_rows_C, nr_cols_C;

    // for simplicity we are going to use square arrays
    nr_rows_A = nr_cols_A = nr_rows_B = nr_cols_B = nr_rows_C = nr_cols_C = 3;
    
    float *h_A = (float *)malloc(nr_rows_A * nr_cols_A * sizeof(float));
    float *h_B = (float *)malloc(nr_rows_B * nr_cols_B * sizeof(float));
    float *h_C = (float *)malloc(nr_rows_C * nr_cols_C * sizeof(float));

    // Allocate 3 arrays on GPU
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A,nr_rows_A * nr_cols_A * sizeof(float));
    hipMalloc(&d_B,nr_rows_B * nr_cols_B * sizeof(float));
    hipMalloc(&d_C,nr_rows_C * nr_cols_C * sizeof(float));

    // ....

    //Free GPU memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);  

    // Free CPU memory
    free(h_A);
    free(h_B);
    free(h_C);    

    return 0;
}
